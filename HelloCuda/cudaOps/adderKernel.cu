#include "hip/hip_runtime.h"
#include "adderKernel.cuh"

__global__ void gpuAdd(int* d_a, int* d_b, int* d_c) {
    *d_c = *d_a + *d_b;
}

int gpuAdd(int const a, int const b) {
    int h_c;
    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    gpuAdd << <1, 1 >> > (d_a, d_b, d_c);

    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return h_c;
}
