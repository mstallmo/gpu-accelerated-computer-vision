#include "hip/hip_runtime.h"
#include "kernel.cuh"

#define N 5

__global__ void gpuSquareKernel(float* d_in, float* d_out)
{
    int tid = threadIdx.x;
    float temp = d_in[tid];
    d_out[tid] = temp * temp;
}

void gpuSquare(float* h_in, float* h_out)
{
    float *d_in, *d_out;

    hipMalloc((void**)&d_in, N * sizeof(float));
    hipMalloc((void**)&d_out, N * sizeof(float));

    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);

    gpuSquareKernel << <1, N >> > (d_in, d_out);

    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_in);
    hipFree(d_out);
}
