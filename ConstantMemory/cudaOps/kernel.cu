#include "hip/hip_runtime.h"
#include "kernel.cuh"

__constant__ int constant_f;
__constant__ int constant_g;
#define N 5

__global__ void constantMemoryGpu(float *d_in, float *d_out) {
  int tid = threadIdx.x;
  d_out[tid] = constant_f * d_in[tid] + constant_g;
}

void constantMemory(float *h_in, float *h_out) {
  float *d_in, *d_out;

  int h_f = 2;
  int h_g = 20;

  hipMalloc((void **)&d_in, sizeof(float) * N);
  hipMalloc((void **)&d_out, sizeof(float) * N);

  hipMemcpy(d_in, h_in, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(constant_f), &h_f, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(constant_g), &h_g, sizeof(int), 0, hipMemcpyHostToDevice);

  constantMemoryGpu<<<1, N>>>(d_in, d_out);
  hipDeviceSynchronize();

  hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
  
  hipFree(d_in);
  hipFree(d_out);
}
