#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <stdio.h>

__global__ void subtractKernel(int d_a, int d_b, int *d_c) { *d_c = d_a - d_b; }

__global__ void multiplyKernel(int *d_a, int *d_b, int *d_c) {
  *d_c = (*d_a) * (*d_b);
}

__global__ void parallelKernel() {
  int bdx = blockIdx.x;
  int tdx = threadIdx.x;

  printf("Block %d; Thread %d\n", bdx, tdx);
}

__global__ void cubeVectorKernel(int *d_in, int *d_out) {
  int tdx = threadIdx.x;
  int temp = d_in[tdx];
  d_out[tdx] = temp * temp * temp;
}

int subtractGpu(int h_a, int h_b) {
  int h_c;
  int *d_c;
  hipMalloc((void **)&d_c, sizeof(int));

  subtractKernel<<<1, 1>>>(h_a, h_b, d_c);

  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_c);

  return h_c;
}

int multiplyGpu(int h_a, int h_b) {
  int h_c;
  int *d_a, *d_b, *d_c;

  hipMalloc((void **)&d_a, sizeof(int));
  hipMalloc((void **)&d_b, sizeof(int));
  hipMalloc((void **)&d_c, sizeof(int));

  hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

  multiplyKernel<<<1, 1>>>(d_a, d_b, d_c);

  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return h_c;
}

void parallel1() {
  printf("10 blocks, 500 threads\n");
  parallelKernel<<<10, 500>>>();
}

void parallel2() {
  printf("5 blocks, 1000 threads\n");
  parallelKernel<<<5, 1000>>>();
}

void parallel3() {
  printf("1000 blocks, 5 threads\n");
  parallelKernel<<<1000, 5>>>();
}

hipDeviceProp_t getDeviceVersion() {
  hipDeviceProp_t device_props;
  hipGetDeviceProperties(&device_props, 0);
  return device_props;
}

void gpuCube(int *h_in, int *h_out) {
  int *d_in, *d_out;

  hipMalloc((void **)&d_in, 50 * sizeof(int));
  hipMalloc((void **)&d_out, 50 * sizeof(int));

  hipMemcpy(d_in, h_in, 50 * sizeof(int), hipMemcpyHostToDevice);

  cubeVectorKernel<<<1, 50>>>(d_in, d_out);

  hipMemcpy(h_out, d_out, 50 * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}
