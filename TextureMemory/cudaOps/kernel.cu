#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <iostream>

#define NUM_THREADS 10
#define N 10

texture<float, 1, hipReadModeElementType> textureRef;

__global__ void textureMemoryGpu(int n, float *d_out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float temp = tex1D(textureRef, float(idx));
    d_out[idx] = temp;
  }
}

void runTextureMemoryGpu() {
  int num_blocks = N / NUM_THREADS + ((N % NUM_THREADS) ? 1 : 0);

  float *d_out;
  hipMalloc((void **)&d_out, N * sizeof(float));
  float *h_out = (float *)malloc(sizeof(float) * N);
  float h_in[N];

  for (int i = 0; i < N; i++) {
    h_in[i] = float(i);
  }

  hipArray *cu_Array;
  hipMallocArray(&cu_Array, &textureRef.channelDesc, N, 1);
  hipMemcpyToArray(cu_Array, 0, 0, h_in, sizeof(float) * N,
                    hipMemcpyHostToDevice);

  hipBindTextureToArray(textureRef, cu_Array);

  textureMemoryGpu<<<num_blocks, NUM_THREADS>>>(N, d_out);

  hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

  std::cout << "Use of Texture memory on GPU" << std::endl;
  for (int i = 0; i < N; i++) {
      std::cout << "Average between the two nearest element is " << h_out[i] << std::endl;
  }

  free(h_out);
  hipFree(d_out);
  hipFreeArray(cu_Array);
  hipUnbindTexture(textureRef);
}
