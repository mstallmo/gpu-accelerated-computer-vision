#include "hip/hip_runtime.h"
#include "kernel.cuh"

#define N 5000

__global__ void gpuAddKernel(int *d_a, int *d_b, int *d_c) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    d_c[tid] = d_a[tid] + d_b[tid];
    tid += blockDim.x * gridDim.x;
  }
}

void gpuAdd(int *h_a, int *h_b, int *h_c) {

  int *d_a, *d_b, *d_c;

  hipMalloc((void **)&d_a, N * sizeof(int));
  hipMalloc((void **)&d_b, N * sizeof(int));
  hipMalloc((void **)&d_c, N * sizeof(int));

  hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

  gpuAddKernel<<<512, 1024>>>(d_a, d_b, d_c);

  hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
